#include "hip/hip_runtime.h"
#include "gpuhandle.h"

__global__ void SetupRandKernel(hiprandState_t *state, unsigned long long seed) 
{
    const unsigned int tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    /* Each thread gets same seed, a different sequence number,
     no offset */
    hiprand_init(seed, tidx, 0, &state[tidx]);
}

void GPUHandle::Init(int dev_id, unsigned int _streamcnt)
{
	streamcnt = _streamcnt;
	hipDeviceReset();
	hipSetDevice(dev_id);
	streams = new hipStream_t[streamcnt];
	for(unsigned int id = 0; id < streamcnt; ++id)
	{
		hipStreamCreate(&streams[id]);
	}
	hipblasCreate(&cublashandle);
	hipsparseCreate(&cusparsehandle);
	hiprandCreateGenerator(&curandgenerator, HIPRAND_RNG_PSEUDO_DEFAULT);
	
	hiprandSetPseudoRandomGeneratorSeed(curandgenerator, time(NULL));
	
    hipMalloc((void **)&devRandStates, NUM_RND_STREAMS * sizeof(hiprandState_t));
	SetupRandKernel<<<NUM_RND_BLOCKS, NUM_RND_THREADS_PER_BLOCK>>>(devRandStates, 1 + time(NULL)*2);
}

void GPUHandle::Destroy()
{
	for(unsigned int id = 0; id < streamcnt; ++id)
	{
		hipStreamDestroy(streams[id]);
	}
	hipblasDestroy(cublashandle);
	hipsparseDestroy(cusparsehandle);
	hiprandDestroyGenerator(curandgenerator);
    hipFree(devRandStates);
	streamcnt = 0U;
}

hiprandState_t* GPUHandle::devRandStates = NULL;
hipStream_t* GPUHandle::streams = NULL;
hipblasHandle_t GPUHandle::cublashandle;
hipsparseHandle_t GPUHandle::cusparsehandle;
hiprandGenerator_t GPUHandle::curandgenerator;
unsigned int GPUHandle::streamcnt = 1U;
